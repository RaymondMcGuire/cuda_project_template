#include "hip/hip_runtime.h"
﻿/*
 * @Author: Xu.Wang 
 * @Date: 2020-04-24 00:06:02 
 * @Last Modified by: Xu.Wang
 * @Last Modified time: 2020-04-24 01:17:50
 */
#include <hip/hip_runtime.h>
#include <cuda_add.h>

__global__ void kernel_add(float *a, float* b,int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i=index;i<n;i+=stride){
        b[i] = a[i] + b[i];
    }
}

void cudaAdd(float *a, float* b,int n)
{   
    float*x;
    float*y;
    hipHostMalloc((void**)&x, n * sizeof(float));
    hipHostMalloc((void**)&y, n * sizeof(float));
    hipMemcpy(x, a, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y, b, n * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    kernel_add<<<numBlocks,blockSize>>>(x, y, n);
    hipDeviceSynchronize();

    hipMemcpy(b, y, n * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(x);
    hipFree(y);
}