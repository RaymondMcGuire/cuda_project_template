#include "hip/hip_runtime.h"
/*
 * @Author: Xu.Wang 
 * @Date: 2020-04-24 00:06:07 
 * @Last Modified by: Xu.Wang
 * @Last Modified time: 2020-04-24 01:14:48
 */
#include <hip/hip_runtime.h>
#include <cuda_double.h>
#include<iostream>

__global__ void kernel_double(int *in, int *out, const int n)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i=index;i<n;i+=stride){
        out[i] = in[i]*2;
    }
}

void cudaDouble(int *hIn, int *hOut,const int n)
{
    int *dIn;
    int *dOut;
    hipHostMalloc((void**)&dIn, n * sizeof(int));
    hipHostMalloc((void**)&dOut, n * sizeof(int));
    hipMemcpy(dIn, hIn, n * sizeof(int), hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    kernel_double<<<numBlocks,blockSize>>>(dIn, dOut, n);
    hipDeviceSynchronize();

    hipMemcpy(hOut, dOut, n * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(dIn);
    hipFree(dOut);
}